
#include <hip/hip_runtime.h>
extern "C" {

__global__ void kernel_vadd(const float *a, const float *b, float *c)
{
    int i = blockIdx.x *blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}


__global__ void kernel_float4(const float4 *a, float4 *b)
{
    b[0] = a[0];
}

__global__ void kernel_uint2(const uint2 *a, uint2 *b)
{
    b[0] = a[0];
}

__global__ void kernel_z(const float4 *a, float *b)
{
    float4 aa = a[0];
    b[0] = aa.x;
    b[1] = aa.y;
    b[2] = aa.z;
    b[3] = aa.w;
}

}
